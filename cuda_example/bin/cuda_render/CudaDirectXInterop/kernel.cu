#include "hip/hip_runtime.h"
﻿
#include "kernel.cuh"

ID3D11Device* d3d11Device = NULL;


void gpuAssert(hipError_t code, const char* file, int line)
{
	if (code != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		fflush(stdout);
	}
}

void setD3D11device(void* device) {
	d3d11Device = reinterpret_cast<ID3D11Device*>(device);
}

void renderToTexture(void* sharedHandle, size_t resourceSize, unsigned int width, unsigned int height) {
	hipExternalMemoryHandleDesc externalHandleDesc;
	memset(&externalHandleDesc, 0, sizeof(externalHandleDesc));

	externalHandleDesc.type = hipExternalMemoryHandleTypeD3D11Resource;
	externalHandleDesc.handle.win32.handle = sharedHandle;
	externalHandleDesc.size = resourceSize;
	externalHandleDesc.flags = hipExternalMemoryDedicated;

	hipExternalMemory_t externalMemory;

	hipError_t err = hipImportExternalMemory(&externalMemory, &externalHandleDesc);
	cudaCheckPrintError(err);

	cudaExternalMemoryMipmappedArrayDesc cuExtmemMipDesc{};
	cuExtmemMipDesc.extent = make_hipExtent(width, height, 0);
	cuExtmemMipDesc.formatDesc = hipCreateChannelDesc<uint4>();
	cuExtmemMipDesc.numLevels = 1;
	cuExtmemMipDesc.flags = hipArraySurfaceLoadStore;

	hipMipmappedArray_t cuMipArray{};

	err = cudaExternalMemoryGetMappedMipmappedArray(&cuMipArray, externalMemory, &cuExtmemMipDesc);
	cudaCheckPrintError(err);

	hipArray_t cuArray{};
	err = hipGetMipmappedArrayLevel(&cuArray, cuMipArray,0);
	cudaCheckPrintError(err);

	hipResourceDesc cuResDesc{};
	cuResDesc.resType = hipResourceTypeArray;
	cuResDesc.res.array.array = cuArray;

	hipSurfaceObject_t cuSurface{};
	err = hipCreateSurfaceObject(&cuSurface, &cuResDesc);
	cudaCheckPrintError(err);

}